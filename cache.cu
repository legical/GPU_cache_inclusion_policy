#include "hip/hip_runtime.h"
#include "util.cu"
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <utility>
#include <thread>

using namespace std;

#define DATATYPE float
#define L1_MAX_SIZE 131072
// 64KB 64 * 1024 = 65536
#define L1_SIZE 65536
// 64KB 64 * 1024 = 65536
#define SHARED_SIZE 65536
#define L2_SIZE 2359296
#define strige 8
// L1_SIZE / sizeof(DATATYPE) = 16384
#define L1_limit 16384
// lock-based
__device__ volatile int g_mutex = 0;

// GPU lock-based synchronization function
__device__ void __gpu_sync(int goalVal)
{
    // thread ID in a block
    int tid_in_block = getThreadIdInBlock();
    // only thread 0 is used for synchronization
    if (tid_in_block == 0)
    {
        atomicAdd((int *)&g_mutex, 1);
        printf("Block %d 's mutex is %d , wish %d .\n", getBlockIDInGrid(), g_mutex, goalVal);
        // only when all blocks add 1 go g_mutex
        // will g_mutex equal to goalVal
        while (g_mutex != goalVal)
        {
            // Do nothing here. Until for synchronization
        }
    }
    __syncthreads();
}

//初始化数组，a[i]=0
template <class T>
void init_order(T *a, int n, int flag)
{
    for (int i = 0; i < n; i++)
    {
        a[i] = (i + strige) * flag;
    }
}

__global__ void cache(int clockRate, DATATYPE *GPU_array_L1, DATATYPE *GPU_array_L2, DATATYPE **dura)
{
    // int array_num = L1_SIZE / sizeof(DATATYPE) / strige + 1;
    uint32_t i = 0;
    uint32_t step = 0;
    __shared__ DATATYPE s_tvalue[L1_limit / strige + 1];
    extern __shared__ DATATYPE s2_tvalue[];
    // __shared__ DATATYPE fence[2];

    uint32_t smid = getSMID();
    uint32_t blockid = getBlockIDInGrid();
    uint32_t threadid = getThreadIdInBlock();
    __syncthreads();
    printf("Blcok %d is running in sm %d.\n", blockid, smid);

    // L1 hit
    i = threadid;
    while (i < L1_limit)
    {
        i = GPU_array_L1[i];
        step++;
        // if (threadid == 0 && blockid == 0)
        // printf("Thread : %d \t step : %d \t i : %d \t Limit is %d\n", threadid, step, i, L1_limit);
    }

    __gpu_sync(2);
    if (threadid == 0)
        printf("block %d test loading L1 cache over.\n", blockid);

    // Load L1 cache
    if (blockid == 0)
    {
        step = 0;
        DATATYPE Start_time = get_time(clockRate);
        for (i = threadid; i < L1_limit;)
        {
            uint32_t index = i;

            i = GPU_array_L1[i];
            step++;
            DATATYPE End_time = get_time(clockRate);
            s_tvalue[index] = End_time - Start_time;
            if (step % 32 == 0)
                printf("First testing L1, %d duration is %.4f\n", index, End_time - Start_time);
        }
        printf("Block 0 first Loading data into L1 cache over.\n");
    }
    // __syncthreads();
    // if (threadid == 0)
    else 
    printf("Block 1 is wating 0's first loading data into L1 cache...\n");
    //等待L1 hit完毕
    // fence[0] += blockid * threadid;
    // __threadfence();
    __gpu_sync(4);

    // Load L2 cache
    if (blockid != 0)
    {
        for (i = threadid; i < L2_SIZE;)
        {

            i = GPU_array_L2[i];
        }
        printf("Block %d loading data into L2 cache over.\n",blockid);
    }
    else
        printf("Block 0 is waiting for 1's Loading data into L2 cache...\n");

    __gpu_sync(6);

    // Load L1 cache again
    if (blockid == 0)
    {
        step = 0;

        DATATYPE Start_time = get_time(clockRate);
        for (i = threadid; i < L1_limit;)
        {
            uint32_t index = i;

            i = GPU_array_L1[i];
            step++;
            DATATYPE End_time = get_time(clockRate);
            s2_tvalue[index] = End_time - Start_time;
            if (step % 32 == 0)
                printf("Second testing L1, %d duration is %.4f\n", index, End_time - Start_time);
        }
        // __syncthreads();
        //保存两次的访问时间
        for (i = threadid; i < L1_limit;)
        {
            dura[0][i] = s_tvalue[i];
            dura[1][i] = s2_tvalue[i];
            i = GPU_array_L1[i];
        }
        if (threadid == 0)
            dura[2][0] = step;
    }
    // __syncthreads();
    __gpu_sync(8);

    //等待L1 load again完毕
    // fence[1] += blockid * threadid;
    // __threadfence();
}

void main_test(int clockRate, DATATYPE *array_L1, DATATYPE *array_L2)
{
    int blocks = 2;
    int threads = 1;
    int dura_num = 3;
    DATATYPE **dura;
    dura = (DATATYPE **)malloc(sizeof(DATATYPE *) * dura_num);
    for (int i = 0; i < dura_num; i++)
    {
        //初始化为0
        dura[i] = (DATATYPE *)malloc(L1_SIZE);
        init_order(dura[i], L1_limit, 0);
    }
    DATATYPE *GPU_array_L1;
    DATATYPE *GPU_array_L2;
    hipMalloc((void **)&GPU_array_L1, L1_SIZE);
    hipMalloc((void **)&GPU_array_L2, sizeof(DATATYPE) * L2_SIZE);
    hipMemcpy(GPU_array_L1, array_L1, L1_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(GPU_array_L2, array_L2, sizeof(DATATYPE) * L2_SIZE, hipMemcpyHostToDevice);
    hipFuncSetAttribute(reinterpret_cast<const void*>(cache), hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_SIZE);
    printf("init shared memory size over.\n");
    // kernel here
    cache<<<blocks, threads, 32 * 1024>>>(clockRate, GPU_array_L1, GPU_array_L2, dura);

    hipDeviceSynchronize();

    //读写文件。文件存在则被截断为零长度，不存在则创建一个新文件
    FILE *fp = fopen("./out/cache.csv", "w+");
    if (fp == NULL)
    {
        fprintf(stderr, "fopen() failed.\n");
        exit(EXIT_FAILURE);
    }
    fprintf(fp, "step,1_L1_duration,2_L1_duration\n");
    for (int i = 0; i < dura[2][0] * threads; i++)
    {
        fprintf(fp, "%d,%.4f,%.4f\n", i, dura[0][i], dura[1][i]);
    }

    fclose(fp);

    hipFree(GPU_array_L1);
    hipFree(GPU_array_L2);
    for (int i = 0; i < dura_num; i++)
    {
        free(dura[i]);
    }
    free(dura);
}

int main()
{
    int device = 0;
    int flag = 1;
    hipDeviceProp_t prop;
    hipSetDevice(device);
    // printf("device:%d\n",device);
    hipGetDeviceProperties(&prop, device);
    int clockRate = prop.clockRate;
    int sm_number = prop.multiProcessorCount;
    printf("*********   This GPU has %d SMs   *********\n", sm_number);
    // output GPU prop

    printf("L1size: %ld \t sizeoftype:%d \t L1limt:%d \t L2size:%d \n", L1_SIZE, sizeof(DATATYPE), L1_limit, L2_SIZE);
    getchar();
    DATATYPE *array_L1;
    DATATYPE *array_L2;
    array_L1 = (DATATYPE *)malloc(L1_SIZE);
    array_L2 = (DATATYPE *)malloc(sizeof(DATATYPE) * L2_SIZE);
    init_order(array_L1, L1_limit, flag);
    init_order(array_L2, L2_SIZE, flag);
    main_test(clockRate, array_L1, array_L2);

    free(array_L1);
    free(array_L2);

    return 0;
}